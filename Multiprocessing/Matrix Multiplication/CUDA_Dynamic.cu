#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include <unistd.h>
#define MAX_SIZE 100 // This will be used only as a fallback if dimensions are not provided

// Function to allocate memory for a matrix dynamically
double** allocate_matrix(int rows, int cols) {
    double** matrix = (double**)malloc(rows * sizeof(double*));
    for (int i = 0; i < rows; i++) {
        matrix[i] = (double*)malloc(cols * sizeof(double));
    }
    return matrix;
}

// Function to free the allocated memory for a matrix
void free_matrix(double** matrix, int rows) {
    for (int i = 0; i < rows; i++) {
        free(matrix[i]);
    }
    free(matrix);
}

// CUDA kernel for matrix multiplication
__global__ void matrix_multiplication_kernel(double* matrixA, double* matrixB, double* result, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        double value = 0;
        for (int k = 0; k < colsA; k++) {
            value += matrixA[row * colsA + k] * matrixB[k * colsB + col];
	    //usleep(1000);
        }
        result[row * colsB + col] = value;
    }
}

// Function to perform matrix multiplication on the GPU
void matrix_multiplication_cuda(double** matrixA, double** matrixB, int rowsA, int colsA, int colsB, double** result) {
    // Allocate device memory
    double *d_matrixA, *d_matrixB, *d_result;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&d_matrixA, rowsA * colsA * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for matrixA: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    hipMalloc((void**)&d_matrixB, colsA * colsB * sizeof(double));
    hipMalloc((void**)&d_result, rowsA * colsB * sizeof(double));

    // Copy input matrices to device
    hipMemcpy(d_matrixA, matrixA[0], rowsA * colsA * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_matrixB, matrixB[0], colsA * colsB * sizeof(double), hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((colsB + blockSize.x - 1) / blockSize.x, (rowsA + blockSize.y - 1) / blockSize.y);

    // Launch the CUDA kernel
    matrix_multiplication_kernel<<<gridSize, blockSize>>>(d_matrixA, d_matrixB, d_result, rowsA, colsA, colsB);
    //usleep(1);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "matrix_multiplication_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // Synchronize CUDA Device
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    // Copy the result back to the host
    cudaStatus = hipMemcpy(result[0], d_result, rowsA * colsB * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for result: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    // Free device memory
    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_result);
}

// Function to save matrix to a file
void save_matrix_to_file(double** matrix, int rows, int cols, const char* filename) {
    FILE *file = fopen(filename, "w");
    if (file == NULL) {
        printf("Error opening file.\n");
        return;
    }

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fprintf(file, "%.10lf ", matrix[i][j]);
        }
        fprintf(file, "\n");
    }

    fclose(file);
}

// Function to read matrix from a file
void read_matrix_from_file(const char* filename, double** matrix, int rows, int cols) {
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        printf("Error opening file.\n");
        return;
    }

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if (fscanf(file, "%lf", &matrix[i][j]) != 1) {
                printf("Error reading matrix from file.\n");
                fclose(file);
                return;
            }
        }
    }

    fclose(file);
}
char *strremove(char *str, const char *sub) {
    size_t len = strlen(sub);
    if (len > 0) {
        char *p = str;
        while ((p = strstr(p, sub)) != NULL) {
            memmove(p, p + len, strlen(p + len) + 1);
        }
    }
    return str;
}

int main(int argc, char *argv[]) {
    int rowsA, colsA, rowsB, colsB;
    char *filename1, *filename2;
    if (argc == 7) {
        rowsA = atoi(argv[1]);
        colsA = atoi(argv[2]);
        rowsB = atoi(argv[3]);
        colsB = atoi(argv[4]);
        filename1 = argv[5];
        filename2 = argv[6];
    } else {
        printf("Usage: %s <rowsA> <colsA> <rowsB> <colsB> <filename1> <filename2>\n", argv[0]);
        return 1;
    }

    if (colsA != rowsB) {
        printf("Matrix multiplication is not possible due to incompatible dimensions.\n");
        return 1;
    }
    // Dynamic allocation of matrices
    double **matrixA = allocate_matrix(rowsA, colsA);
    double **matrixB = allocate_matrix(rowsB, colsB);
    double **result = allocate_matrix(rowsA, colsB);

    // Read matrices from files (assuming files exist and have correct format)
    read_matrix_from_file(filename1, matrixA, rowsA, colsA);
    read_matrix_from_file(filename2, matrixB, rowsB, colsB);

    // Perform multiplication on the GPU
    clock_t start = clock();
    matrix_multiplication_cuda(matrixA, matrixB, rowsA, colsA, colsB, result);
    clock_t end = clock();

    int cpu_time_used = ((1000)*(end - start)) / CLOCKS_PER_SEC;
    printf("%d\n", cpu_time_used);

    // Save the result to a file
    save_matrix_to_file(result, rowsA, colsB, "matrixC_CUDA.txt");

    // Freeing dynamically allocated memory
    //free_matrix(matrixA, rowsA);
    //free_matrix(matrixB, rowsB);
    //free_matrix(result, rowsA);

    return cpu_time_used;
}
